
#include <hip/hip_runtime.h>
﻿extern "C" {

    __global__ void Sum(int N, const double* __restrict left, const double* __restrict right, double* __restrict output) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			output[i] = left[i] + right[i];
	}

    __global__ void UpdateAdamOptimizer(int N, double beta1, double beta2, double epsilon, double multiplicative_factor,
				const double* __restrict dW, double* __restrict W,
				double* __restrict adam_vW, double* __restrict adam_sW) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			double dw = dW[i];
			adam_vW[i] = beta1*adam_vW[i]+(1-beta1)*dw;
            adam_sW[i] = beta2*adam_sW[i]+(1-beta2)*dw*dw;
			W[i] -= (multiplicative_factor * adam_vW[i]) / (sqrt(adam_sW[i]) + epsilon);
		}
	}

	__global__ void ComputeAccuracy(int N, int categoryCount, double *countOk, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted) 	{
		int i = blockIdx.x * blockDim.x + threadIdx.x; 
		if (i < N) {
			if (categoryCount == 1)
			{
				float error = fabsf(yExpectedOneHot[i] - yPredicted[i]);
				countOk[i] = (error < 0.5) ? 1.0 : 0.0;
				return;
			}

			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndexPredicted = startIndex;
			int maxIndexExpected = startIndex;
			for (int j = startIndex+1; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndexPredicted])
					maxIndexPredicted = j;
				if (yExpectedOneHot[j] > yExpectedOneHot[maxIndexExpected])
					maxIndexExpected = j;
			}
			countOk[i] = (maxIndexPredicted == maxIndexExpected) ? 1.0f : 0.0f;
		}
	}

	__global__ void ComputeCategoricalCrossentropyLoss(int N, int categoryCount, double *losses, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			double loss = 0.0;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndex = startIndex;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				double predicted = yPredicted[j];
				double expected = yExpectedOneHot[j];
				if (predicted > 0)
					loss -= expected * logf(predicted);
			}
			losses[i] = loss;
		}
	}

	__global__ void ComputeBinaryCrossentropyLoss(int N, int categoryCount, double *losses, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			double loss = 0.0;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndex = startIndex;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				double predicted = yPredicted[j];
				double expected = yExpectedOneHot[j];
				if ((predicted > 0.0)&&(predicted<1.0))
					loss -= (expected*logf(predicted) + (1.0-expected)*logf(1.0-predicted))/ categoryCount;
			}
			losses[i] = loss;
		}
	}

	__global__ void Concatenate(int N, int m, double* __restrict concat, int concatMultDim0, const double* __restrict a, int aMultDim0, const double* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		concat[i] = (colInConcat<aMultDim0)?a[row*aMultDim0+colInConcat]:b[row*bMultDim0+colInConcat-aMultDim0];
	}

	__global__ void Split(int N, int m, const double* __restrict concat, int concatMultDim0, double* __restrict a, int aMultDim0, double* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
	}
}
