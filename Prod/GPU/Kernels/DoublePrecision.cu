
#include <hip/hip_runtime.h>
﻿extern "C" {

    __global__ void Sum(int N, const double* __restrict left, const double* __restrict right, double* __restrict output) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			output[i] = left[i] + right[i];
	}

    __global__ void UpdateAdamOptimizer(int N, double beta1, double beta2, double epsilon, double multiplicative_factor,
				const double* __restrict dW, double* __restrict W,
				double* __restrict adam_vW, double* __restrict adam_sW) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			double dw = dW[i];
			adam_vW[i] = beta1*adam_vW[i]+(1-beta1)*dw;
            adam_sW[i] = beta2*adam_sW[i]+(1-beta2)*dw*dw;
			W[i] -= (multiplicative_factor * adam_vW[i]) / (sqrt(adam_sW[i]) + epsilon);
		}
	}

	//TODO remove this function
	__global__ void UpdateSGDOptimizer(int N, double learningRate, double momentum, double decay, bool usenesterov,
		const double* __restrict dW, double* __restrict W, double* __restrict velocity) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			double dw = dW[i];
			velocity[i] = (momentum * velocity[i]) - (dw * learningRate);
			if (usenesterov)
			{
				W[i] += momentum * velocity[i] - (dw * learningRate);
			}
			else
			{
				W[i] += velocity[i];
			}

		}
	}

	__global__ void ComputeAccuracy(int N, int categoryCount, double *countOk, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted) 	{
		int i = blockIdx.x * blockDim.x + threadIdx.x; 
		if (i < N) {
			if (categoryCount == 1)
			{
				float error = fabsf(yExpectedOneHot[i] - yPredicted[i]);
				countOk[i] = (error < 0.5) ? 1.0 : 0.0;
				return;
			}

			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndex = startIndex;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndex])
					maxIndex = j;
			}
			countOk[i] = (yExpectedOneHot[maxIndex] > 0.9) ? 1.0 : 0.0;
		}
	}

	__global__ void ComputeCategoricalCrossentropyLoss(int N, int categoryCount, double *losses, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			double loss = 0.0;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndex = startIndex;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				double predicted = yPredicted[j];
				double expected = yExpectedOneHot[j];
				if (predicted > 0)
					loss -= expected * logf(predicted);
			}
			losses[i] = loss;
		}
	}

	__global__ void ComputeBinaryCrossentropyLoss(int N, int categoryCount, double *losses, const double* __restrict yExpectedOneHot, const double* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			double loss = 0.0;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndex = startIndex;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				double predicted = yPredicted[j];
				double expected = yExpectedOneHot[j];
				if ((predicted > 0.0)&&(predicted<1.0))
					loss -= (expected*logf(predicted) + (1.0-expected)*logf(1.0-predicted))/ categoryCount;
			}
			losses[i] = loss;
		}
	}

	
}