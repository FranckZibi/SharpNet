
#include <hip/hip_runtime.h>
﻿extern "C" {

    __global__ void Sum(int N, const float* __restrict left, const float* __restrict right, float* __restrict output) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			output[i] = left[i] + right[i];
	}

    __global__ void UpdateAdamOptimizer(int N, float beta1, float beta2, float epsilon, float multiplicative_factor,
				const float* __restrict dW, float* __restrict W,
				float* __restrict adam_vW, float* __restrict adam_sW) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			float dw = dW[i];
			adam_vW[i] = beta1*adam_vW[i]+(1-beta1)*dw;
            adam_sW[i] = beta2*adam_sW[i]+(1-beta2)*dw*dw;
			W[i] -= (multiplicative_factor * adam_vW[i]) / (sqrtf(adam_sW[i]) + epsilon);
		}
	}

	__global__ void ComputeAccuracy(int N, int categoryCount, float *countOk, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			if (categoryCount == 1)
			{
				float error = fabsf(yExpectedOneHot[i] - yPredicted[i]);
				countOk[i] = (error < 0.5f) ? 1.0f : 0.0f;
				return;
			}

			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndexPredicted = startIndex;
			int maxIndexExpected = startIndex;
			for (int j = startIndex+1; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndexPredicted])
					maxIndexPredicted = j;
				if (yExpectedOneHot[j] > yExpectedOneHot[maxIndexExpected])
					maxIndexExpected = j;
			}
			countOk[i] = (maxIndexPredicted == maxIndexExpected) ? 1.0f : 0.0f;
		}
	}

	__global__ void MultiplyEachRowIntoSingleValue(int nbRows, int nbCols, float *result, const float* __restrict a, const float* __restrict b) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows) {
			a += row*nbCols;
			b += row*nbCols;
			float sumInRow = 0;
			for(int i=0;i<nbCols;++i)
			{
				sumInRow += (*a)*(*b);
				++a;
				++b;
			}
			result[row] = sumInRow;
		}
	}

	__global__ void ComputeCategoricalCrossentropyLoss(int N, int categoryCount, float *losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				if (predicted > 0)
					loss -= expected * logf(predicted);
			}
			losses[i] = loss;
		}
	}

	__global__ void ComputeBinaryCrossentropyLoss(int N, int categoryCount, float *losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				//if ((predicted>0.01)&&(predicted<0.99f))
				if ((predicted>0.0f)&&(predicted<1.0f))
					loss -= (expected*logf(predicted) + (1.0f-expected)*logf(1.0f-predicted))/ categoryCount;
			}
			losses[i] = loss;
		}
	}




	__global__ void ComputeAccuracyFromCategoryIndexes(int N, int categoryCount, float *countOk, const int* __restrict categoryIndexes, const float* __restrict yPredicted) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int categoryIndex = categoryIndexes[i]; /* the expected category index for element at index 'i' */
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndexPredicted = startIndex;
			for (int j = startIndex+1; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndexPredicted])
					maxIndexPredicted = j;
			}
			countOk[i] = ( (maxIndexPredicted-startIndex) == categoryIndex) ? 1.0f : 0.0f;
		}
	}

	__global__ void ComputeCategoricalCrossentropyLossFromCategoryIndexes(int N, int categoryCount, float *losses, const int* __restrict categoryIndexes, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int categoryIndex = categoryIndexes[i]; /* the expected category index for element at index 'i' */
			int startIndex = i * categoryCount;
			float predictedForExpectedCategory = yPredicted[startIndex+categoryIndex];
			if (predictedForExpectedCategory > 0)
				losses[i] = -logf(predictedForExpectedCategory);
			else
				losses[i] = 0.0f;
		}
	}

	__global__ void ComputeBinaryCrossentropyLossFromCategoryIndexes(int N, int categoryCount, float *losses, const int* __restrict categoryIndexes, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int categoryIndex = categoryIndexes[i]; /* the expected category index for element at index 'i' */
			int startIndex = i * categoryCount;
			float loss = 0.0f;
			for (int category = 0; category < categoryCount; ++category)
			{
				float predicted = yPredicted[startIndex+category];
				float error = (category == categoryIndex)? predicted : (1.0f-predicted);
				if (error > 0)
				{
					loss -= logf(error);
				}
			}
			losses[i] = loss/ categoryCount;
		}
	}

	__global__ void Concatenate(int N, int m, float* __restrict concat, int concatMultDim0, const float* __restrict a, int aMultDim0, const float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		concat[i] = (colInConcat<aMultDim0)?a[row*aMultDim0+colInConcat]:b[row*bMultDim0+colInConcat-aMultDim0];
	}

	__global__ void Split(int N, int m, const float* __restrict concat, int concatMultDim0, float* __restrict a, int aMultDim0, float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
	}
}
