
#include <hip/hip_runtime.h>
﻿extern "C" {

	__device__ inline float sigmoidf(float x) {
		return 1.0f / (1 + expf(-x));
	}

    __global__ void Sum(int N, const float* __restrict left, const float* __restrict right, float* __restrict output) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			output[i] = left[i] + right[i];
	}

    __global__ void UpdateAdamOptimizer(int N, float beta1, float beta2, float epsilon, float multiplicative_factor,
				const float* __restrict dW, float* __restrict W,
				float* __restrict adam_vW, float* __restrict adam_sW) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			float dw = dW[i];
			adam_vW[i] = beta1*adam_vW[i]+(1-beta1)*dw;
            adam_sW[i] = beta2*adam_sW[i]+(1-beta2)*dw*dw;
			W[i] -= (multiplicative_factor * adam_vW[i]) / (sqrtf(adam_sW[i]) + epsilon);
		}
	}

	__global__ void ComputeAccuracy(int N, int categoryCount, float *countOk, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			if (categoryCount == 1)
			{
				float error = fabsf(yExpectedOneHot[i] - yPredicted[i]);
				countOk[i] = (error < 0.5f) ? 1.0f : 0.0f;
				return;
			}

			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndexPredicted = startIndex;
			int maxIndexExpected = startIndex;
			for (int j = startIndex+1; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndexPredicted])
					maxIndexPredicted = j;
				if (yExpectedOneHot[j] > yExpectedOneHot[maxIndexExpected])
					maxIndexExpected = j;
			}
			countOk[i] = (maxIndexPredicted == maxIndexExpected) ? 1.0f : 0.0f;
		}
	}


	__device__  bool IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(const float* __restrict expected, const float* __restrict predicted, int endIndexExcluded, int *pNexIndexToCheck, int subCategoriesCount)
	{
		int subCategoriesFound = 0;
		int predictedSubCategoryId = -1;
		float bestPredictedSubCategoryProba = -1.0f;
		int expectedSubCategoryId = -1;
		float bestExpectedSubCategoryProba = -1.0f;
		bool isAccurate = true;
		bool previousIndexWasProba = false;

		while (subCategoriesFound < subCategoriesCount && (*pNexIndexToCheck < endIndexExcluded))
		{
			float expectedProba = expected[*pNexIndexToCheck];
			float predictedProba = predicted[*pNexIndexToCheck];
			if (fabsf(expectedProba) < 9.5f)
			{
				previousIndexWasProba = true;
				++subCategoriesFound;
				if (expectedProba > bestExpectedSubCategoryProba)
				{
					bestExpectedSubCategoryProba = expectedProba;
					expectedSubCategoryId = subCategoriesFound - 1;
				}
				if (predictedProba > bestPredictedSubCategoryProba)
				{
					bestPredictedSubCategoryProba = predictedProba;
					predictedSubCategoryId = subCategoriesFound - 1;
				}
				*pNexIndexToCheck += 1;
			}
			else
			{
				int count = (int)(fabsf(expectedProba) + 0.5f) / 10;
				if (expectedProba < 0)
				{
					//we need to skip 'count' indexes
					*pNexIndexToCheck += count;
				}
				else
				{
					*pNexIndexToCheck += 1;
					bool subCategoryIsAccurate = IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(expected, predicted, endIndexExcluded, pNexIndexToCheck, count);
					isAccurate = subCategoryIsAccurate && isAccurate;
				}
				if (!previousIndexWasProba)
				{
					++subCategoriesFound;
				}
				previousIndexWasProba = false;
			}
		}
		return (expectedSubCategoryId == predictedSubCategoryId) && isAccurate;
	}

	__device__ inline float IsCountAssociateWithAboveProba(float f) { return f > 5.0f && ((int)(f + 0.1f)) % 10 == 1; }
	__device__ inline float IsProba(float f) { return fabsf(f) < 5.0f; }
	__device__ inline float ExtractCount(float f) { return (int)(fabsf(f) + 0.5f) / 10; }

	__global__ void ComputeSingleAccuracyForCategoricalCrossentropyWithHierarchy(int N, int nbCols, float* countOk, const float* __restrict expected, const float* __restrict predicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int nexIndexToCheck = 0;
			countOk[i] = IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(expected + i * nbCols, predicted + i * nbCols, nbCols, &nexIndexToCheck, 10000000)
				? 1.0f 
				: 0.0f;
		}
	}

	__device__  void SoftmaxWithHierarchy(const float* activationParameter, float* y, int endIndexExcluded, int* pNexIndexToCheck)
	{
		float param = activationParameter[*pNexIndexToCheck];
		y[*pNexIndexToCheck] = param;
		int subCategoriesCount = ExtractCount(param);
		*pNexIndexToCheck += 1;
		
		//we only allocate an array if we have more then '10' elements
		int smallIntArray[10];
		int* indexesProba = (subCategoriesCount > 10) ? (int*)malloc(subCategoriesCount * sizeof(int)) : (&smallIntArray[0]);

		float maxProba = -1e9f;
		bool probaFound = false;

		for (int subCategoriesFound = 0; subCategoriesFound < subCategoriesCount; ++subCategoriesFound)
		{
			float expectedProba = activationParameter[*pNexIndexToCheck];
			if (IsProba(expectedProba))
			{
				maxProba = fmaxf(maxProba, y[*pNexIndexToCheck]);
				indexesProba[subCategoriesFound] = *pNexIndexToCheck;
				probaFound = true;
				*pNexIndexToCheck += 1;
				if (*pNexIndexToCheck < endIndexExcluded && IsCountAssociateWithAboveProba(activationParameter[*pNexIndexToCheck]))
				{
					SoftmaxWithHierarchy(activationParameter, y, endIndexExcluded, pNexIndexToCheck);
				}
			}
			else
			{
				SoftmaxWithHierarchy(activationParameter, y, endIndexExcluded, pNexIndexToCheck);
			}
		}

		if (probaFound)
		{
			float sumExp = 0.0f;
			for (int i = 0; i < subCategoriesCount; ++i)
			{
				int idx = indexesProba[i];
				float tmp = expf(y[idx] - maxProba);
				sumExp += tmp;
				y[idx] = tmp;
			}
			for (int i = 0; i < subCategoriesCount; ++i)
			{
				y[indexesProba[i]] /= sumExp;
			}
		}

		if (subCategoriesCount > 10)
		{
			free(indexesProba);
		}
	}

	__global__ void ComputeSoftmaxWithHierarchy(int N, int nbCols, const float* activationParameter, float* y)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int nexIndexToCheck = 0;
			SoftmaxWithHierarchy(activationParameter, y + i * nbCols, nbCols, &nexIndexToCheck);
		}
	}

	__global__ void ComputeSoftmaxGradientWitHierarchy(int N, int nbCols, const float* activationParameter, const float* y, const float* dy, float* dx)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float expectedProba = activationParameter[i%nbCols];
			if (IsProba(expectedProba))
			{
				float dyi = dy[i];
				float yi = y[i];
				dx[i] = (fabsf(dyi - 1.0f) < 1e-6) ? (yi * (1 - yi)) : (-yi * dyi);
			}
			else
			{
				dx[i] = expectedProba;
			}
		}
	}

	__global__ void SwishGradient(int N, const float* __restrict Y, const float* __restrict dY, const float* __restrict X, float *dX) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < N) {
			float x = X[row];
		    float sigmoid_x = (fabs(x) < 0.0001f) ? 0.5f : Y[row] / x;
            dX[row] = dY[row] * (sigmoid_x + x * sigmoid_x * (1 - sigmoid_x));
		}
	}
    
	__global__ void MultiplyEachRowIntoSingleValue(int nbRows, int nbCols, float *result, const float* __restrict a, const float* __restrict b) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows) {
			a += row*nbCols;
			b += row*nbCols;
			float sumInRow = 0;
			for(int i=0;i<nbCols;++i)
			{
				sumInRow += (*a)*(*b);
				++a;
				++b;
			}
			result[row] = sumInRow;
		}
	}

	// src tensor (unpadded tensor) has shape (n, c, h_src, w_src)
	// dest tensor (padded tensor) has shape (n, c, h_dest, w_dest) with:
    //		h_dest = top_pad + h_src + bottom_pad;
    //      w_dest = left_pad + w_src + right_pad;
	// N = n*c*h_src = number of distinct rows in 'src' tensor
	__global__ void ApplyZeroPaddingForRowId(int N, int h_src, int w_src, int top_pad, int bottom_pad, int left_pad, int right_pad, float* paddedTensor, float* unpaddedTensor, bool isUnpadding) 
	{
		// 'rowId' is the index of the row in 'src' tensor (0 <= rowId < N with N=n*c*h_src)
		int rowId = blockIdx.x * blockDim.x + threadIdx.x;
		if (rowId < N) {
			//we'll copy the row 'rowId' from 'src' tensor (n, c, h_src, w_src) to 'dest' tensor (n, c, h_dest, w_dest)
            int h_dest = top_pad + h_src + bottom_pad;
            int w_dest = left_pad + w_src + right_pad;
            int row_in = (rowId % h_src);
            int destRowIdx = ((rowId / h_src) * h_dest + row_in + top_pad) * w_dest + left_pad;
            int rowIdx = rowId * w_src;
			if (isUnpadding)
				memcpy(unpaddedTensor+rowIdx, paddedTensor+destRowIdx, sizeof(float)*w_src);
			else
				memcpy(paddedTensor+destRowIdx, unpaddedTensor+rowIdx, sizeof(float)*w_src);
		}
	}

	//'y' shape :               (batchSize, embeddingDim, maxWordCountBySentence)
	//'x' shape:                (batchSize, maxWordCountBySentence)
	//'wordEmbedding' shape:    (vocabularySize, embeddingDim)
	__global__ void WordEmbeddingForwardPropagation(int N, int batchSize, int maxWordCountBySentence, int embeddingDim, int vocabularySize, float* y, float* x, float* wordEmbedding)
	{
		int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (xIndex >= N) return;
		int xRow = xIndex / maxWordCountBySentence; //sentenceId in [0, batchSize-1]
		int xCol = xIndex % maxWordCountBySentence; //word position in sentence, in [0, maxWordCountBySentence-1]
		int wordIndex = (int)(x[xIndex] + 0.1f);	//in [0, vocabularySize-1]
		int indexInWordEmbedding = wordIndex* embeddingDim;
		int indexInY = xRow*(embeddingDim*maxWordCountBySentence)+ xCol;
		for (int embeddingId = 0; embeddingId < embeddingDim; ++embeddingId)
		{
			y[indexInY] = wordEmbedding[indexInWordEmbedding];
			indexInY += maxWordCountBySentence;
			++indexInWordEmbedding;
		}
	}

	//'dw' shape:				(VocabularySize, EmbeddingDim)
	// x shape :                (batchSize,  maxWordCountBySentence)
	// dy shape :               (batchSize, EmbeddingDim,  maxWordCountBySentence)
	__global__ void WordEmbeddingBackwardPropagation(int N, int batchSize, int maxWordCountBySentence, int embeddingDim, int vocabularySize, float* dw, float* x, float* dy)
	{
		int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (xIndex >= N) return;
		int xRow = xIndex / maxWordCountBySentence; //sentenceId, in [0, batchSize-1]
		int xCol = xIndex % maxWordCountBySentence; //word position in sentence in [0, maxWordCountBySentence-1]
		int wordIndex = (int)(x[xIndex] + 0.1f);	//in [0, vocabularySize-1]
		int dwIndex = embeddingDim * wordIndex;
		int dyIndex = xRow * (maxWordCountBySentence * embeddingDim) + xCol;
		for (int embeddingId = 0; embeddingId < embeddingDim; ++embeddingId)
		{
			float valueToAdd = dy[dyIndex];
			atomicAdd(dw+dwIndex, valueToAdd);
			++dwIndex;
			dyIndex += maxWordCountBySentence;
		}
	}

	__global__ void YOLOV3Forward(int N, float* y, float* x, int x_c, int x_h, int x_w, int inputImageHeight, int inputImageWidth, int anchor0Width, int anchor0Height, int anchor1Width, int anchor1Height, int anchor2Width, int anchor2Height) 
	{
		int xpredictionIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (xpredictionIndex < N) {

			int nbAnchors = 3;
			int predictionLength = x_c/nbAnchors;
            int categories = predictionLength - 5;
            int rowStride = inputImageHeight / x_h;
            int colStride = inputImageWidth / x_w;
			int xpredictionIndexBackup = xpredictionIndex;
			int elementId = xpredictionIndex / (nbAnchors*x_h*x_w);
			xpredictionIndex = xpredictionIndex %(nbAnchors*x_h*x_w);
			int boxId = xpredictionIndex / (x_h*x_w);
			xpredictionIndex = xpredictionIndex %(x_h*x_w);
			int x_row = xpredictionIndex / (x_w);
			int x_col = xpredictionIndex %(x_w);

			xpredictionIndex= xpredictionIndexBackup;
			int xIndex = elementId*x_c*x_h*x_w + boxId*predictionLength*x_h*x_w + x_row*x_w  + x_col;
			int yIndex = elementId*x_c*x_h*x_w + x_row*x_c*x_w + x_col*x_c + boxId*predictionLength;

            //box center
            y[yIndex++] = (x_col + sigmoidf(x[xIndex])) * colStride;
            xIndex += x_h*x_w;
            y[yIndex++] = (x_row + sigmoidf(x[xIndex])) * rowStride;
            xIndex += x_h*x_w;

            //box size
            int anchorWidth = (boxId == 0) ? anchor0Width : ((boxId == 1) ? anchor1Width : anchor2Width);
            y[yIndex++] = anchorWidth * expf(x[xIndex]);
            xIndex += x_h*x_w;
            int anchorHeight = (boxId == 0) ? anchor0Height : ((boxId == 1) ? anchor1Height : anchor2Height);
            y[yIndex++] = anchorHeight * expf(x[xIndex]);
            xIndex += x_h*x_w;

            //box confidence
            y[yIndex++] = sigmoidf(x[xIndex]);
            xIndex += x_h*x_w;

            //categories
            for (int i = 0; i < categories; ++i)
            {
                y[yIndex++] = sigmoidf(x[xIndex]);
                xIndex += x_h*x_w;
            }
		}
	}


	// src tensor (tensor before up sampling) has shape (n, c, h_src, w_src)
	// dest tensor (tensor after upsampling) has shape (n, c, rowFactor*h_src, colFactor*w_dest)
	// isUpscaling : true if we are up sampling (from 'src' to 'dest') / false if we are down sampling (from 'dest' to 'src')
	__global__ void UpSampling2D(int N, int channels, int h_src, int w_src, int rowFactor, int colFactor, float* src, float* dest, bool isUpscaling) 
	{
		int srcIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (srcIndex < N) {
			int h_dest = h_src * rowFactor;
			int w_dest = w_src * colFactor;
			float originalElement = src[srcIndex];
			int srcIndexbackup = srcIndex;
	
			int elementId = srcIndex / (channels*h_src*w_src);
			srcIndex = srcIndex %(channels*h_src*w_src);
			int channel = srcIndex / (h_src*w_src);
			srcIndex = srcIndex %(h_src*w_src);
			int row_src = srcIndex / (w_src);
			int col_src = srcIndex %(w_src);
			srcIndex = srcIndexbackup;
			float sum = 0; //only used when down sampling (isUpscaling = false)

			int startOfRow = elementId*(channels*h_dest*w_dest)+channel*(h_dest*w_dest)+ row_src*rowFactor *w_dest + col_src* colFactor;
			for(int rowOffset=0;rowOffset<rowFactor;++rowOffset)
			{
				int idx_dest = startOfRow;
				for(int colOffset=0;colOffset<colFactor;++colOffset)
				{
					if (isUpscaling)
						dest[idx_dest] = originalElement;
					else
						sum += dest[idx_dest];
					++idx_dest;
				}
				startOfRow += w_dest;
			}
			if (!isUpscaling)
				src[srcIndex] = sum;
		}
	}

	__global__ void ComputeCategoricalCrossentropyLoss(int N, int categoryCount, float *losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				if (predicted > 0)
					loss -= expected * logf(predicted);
			}
			losses[i] = loss;
		}
	}

	__global__ void ComputeHuberLoss(int N, int categoryCount, float huberDelta, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpected[j];
				float diff = expected - predicted;
				if (fabsf(diff) <= huberDelta)
					loss += 0.5f * diff * diff;
				else
					loss += huberDelta*fabs(diff)-0.5f* huberDelta * huberDelta;
			}
			losses[i] = loss;
		}
	}


	__global__ void ComputeLossForCategoricalCrossentropyWithHierarchy(int N, int nbCols, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * nbCols;
			int endIndexExcluded = startIndex + nbCols;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float expected = yExpected[j];
				if (fabsf(expected) < 9.5f)
				{
					if (expected > 1e-6f)
					{
						//expected contains a proba between 0 and 1
						float predicted = yPredicted[j];
						loss += expected * logf(fmaxf(1e-6f, predicted));
					}
				}
				else
				{
					if (expected < 0) 
					{
						//expected contains a description : there is no associated loss
						int count = (int)(fabsf(expected) + 0.5f) / 10;
						//we need to skip 'count' indexes
						j += count - 1; //-1 because the for(;;) loop will also increment 'j'
					}
				}
			}
			losses[i] = -loss;
		}
	}

	__global__ void ComputeBackwardPropagationLossCategoricalCrossentropyWithHierarchy(int N, int nbCols, float* loss, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int startIndex = i * nbCols;
			int endIndexExcluded = startIndex + nbCols;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float expected = yExpected[j];
				if (fabsf(expected) < 9.5f)
				{
					//expected contains a proba between 0 and 1
					loss[j] = yPredicted[j]- expected;
				}
				else
				{
					if (expected < 0)
					{
						//expected contains a number of element to skip: there is no associated loss
						int count = (int)(fabsf(expected) + 0.5f) / 10;
						//we need to skip 'count' indexes
						j += count - 1; //-1 because the for(;;) loop will also increment 'j'
					}
				}
			}
		}
	}

	__global__ void ComputeBackwardPropagationLossHuber(int N, int nbCols, float huberDelta,  float* loss, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int startIndex = i * nbCols;
			int endIndexExcluded = startIndex + nbCols;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yPredicted[j] - yExpected[j];
				loss[j] = fmaxf(fminf(diff, huberDelta), -huberDelta);
			}
		}
	}

	__global__ void ComputeBinaryCrossentropyLoss(int N, int categoryCount, float *losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				//if ((predicted>0.01)&&(predicted<0.99f))
				if ((predicted>0.0f)&&(predicted<1.0f))
					loss -= (expected*logf(predicted) + (1.0f-expected)*logf(1.0f-predicted))/ categoryCount;
			}
			losses[i] = loss;
		}
	}

	__global__ void Concatenate(int N, int m, float* __restrict concat, int concatMultDim0, const float* __restrict a, int aMultDim0, const float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			concat[i] = a[row*aMultDim0+colInConcat];
		else
			concat[i] = b[row*bMultDim0+colInConcat-aMultDim0];

	}

	__global__ void Concatenate3(int N, int m, float* __restrict concat, int concatMultDim0, const float* __restrict a, int aMultDim0, const float* __restrict b, int bMultDim0, const float* __restrict c, int cMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			concat[i] = a[row*aMultDim0+colInConcat];
		else
			concat[i] = (colInConcat<(aMultDim0+bMultDim0))?b[row*bMultDim0+colInConcat-aMultDim0]:c[row*cMultDim0+colInConcat-aMultDim0-bMultDim0];
	}


	__global__ void Split(int N, int m, const float* __restrict concat, int concatMultDim0, float* __restrict a, int aMultDim0, float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
	}

	__global__ void Split3(int N, int m, const float* __restrict concat, int concatMultDim0, float* __restrict a, int aMultDim0, float* __restrict b, int bMultDim0, float* __restrict c, int cMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else if (colInConcat<(aMultDim0+bMultDim0))
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
		else
			c[row*cMultDim0+colInConcat-aMultDim0-bMultDim0] = concat[i];
	}
}

