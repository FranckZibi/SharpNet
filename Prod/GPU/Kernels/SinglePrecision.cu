
#include <hip/hip_runtime.h>
﻿extern "C" {

	__device__ inline float sigmoidf(float x) {
		return 1.0f / (1 + expf(-x));
	}

    __global__ void Sum(int N, const float* __restrict left, const float* __restrict right, float* __restrict output) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
			output[i] = left[i] + right[i];
	}

    __global__ void UpdateAdamOptimizer(int N, float beta1, float beta2, float epsilon, float adamW_l2Regularization, float multiplicative_factor,
				const float* __restrict dW, float* __restrict W,
				float* __restrict adam_vW, float* __restrict adam_sW) {
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
			float dw = dW[i];
			adam_vW[i] = beta1*adam_vW[i]+(1-beta1)*dw;
            adam_sW[i] = beta2*adam_sW[i]+(1-beta2)*dw*dw;
			W[i] -= (multiplicative_factor * adam_vW[i]) / (sqrtf(adam_sW[i]) + epsilon) + adamW_l2Regularization * W[i];
		}
	}


	// for each row of tensor 'x' (of shape (N, cols), normalize the value
	//  y = (x-mean)/volatility
	__global__ void StandardizeInPlaceByRow(int N, int cols, float* __restrict x, float* __restrict mean, float* __restrict variance, float epsilon) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < N) {
			float row_mean = mean[row];
            float row_variance = variance[row];
            int startIndex = row * cols;
            int endIndex = startIndex + cols - 1;
            for (int i = startIndex; i <= endIndex; ++i)
            {
                x[i] = (x[i] - row_mean) / sqrtf(row_variance + epsilon);
            }
		}
	}

	__global__ void BroadcastRowByRow(int rows, int cols, float* __restrict x, float* __restrict col_multiplier, float mult_to_col_multiplier,  float* __restrict col_adder, float mult_to_col_adder, float constant_to_add) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < rows) {

			int xIndex = row * cols;
	        for (int col = 0; col < cols; ++col)
	        {
				float multiplier = (col_multiplier == NULL) ? 1.0f : col_multiplier[col];
		        multiplier *= mult_to_col_multiplier;
		        float adder = (col_adder == NULL) ? 0.0f : col_adder[col];
		        adder = mult_to_col_adder*adder+constant_to_add;
	            x[xIndex] = multiplier * x[xIndex] + adder;
	            ++xIndex;
	        }
		}
	}

	__global__ void BroadcastColByCol(int cols, int rows, float* __restrict x, float* __restrict row_multiplier, float mult_to_row_multiplier,  float* __restrict row_adder, float mult_to_row_adder, float constant_to_add) 
	{
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		if (col < cols) {

			int xIndex = col;
	        for (int row = 0; row < rows; ++row)
	        {
				float multiplier = (row_multiplier == NULL) ? 1.0f : row_multiplier[row];
		        multiplier *= mult_to_row_multiplier;
		        float adder = (row_adder == NULL) ? 0.0f : row_adder[row];
		        adder = mult_to_row_adder*adder+constant_to_add;
	            x[xIndex] = multiplier * x[xIndex] + adder;
	            xIndex += cols;
	        }
		}
	}


	__global__ void numpy_sum_ColByCol(int cols, int rows, const float* __restrict x, float* __restrict sum_buffer) 
	{
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		if (col < cols) {

			float col_sum = 0.0f;
            for (int row = 0; row< rows; ++row)
            {
				col_sum += x[row * cols + col];
            }
			sum_buffer[col] = col_sum;
		}
	}

	__global__ void numpy_sum_RowByRow(int rows, int cols, const float* __restrict x, float* __restrict sum_buffer) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < rows) {

			float row_sum = 0.0f;
			int index = row * cols;
            for (int col = 0; col< cols; ++col)
            {
				row_sum += x[index];
				++index;
            }
			sum_buffer[row] = row_sum;
		}
	}

	__global__ void LayerNormalizationBackward(int rows, int cols, const float* __restrict x,  const float* __restrict dy,  float* __restrict dx,  const float* __restrict gammas,  const float* __restrict mean,  const float* __restrict variance, float epsilon) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < rows) {
			float mean_row = mean[row];
			float variance_row = variance[row];
			float volatility_row = sqrtf(variance_row + epsilon);
			float dvariance_row = 0;
			float dmean_row = 0;
			int index =row*cols;
			for (int col = 0; col < cols; ++col)
			{
			    float tmp0 = (dy[index+col] * gammas[col]);
			    dvariance_row += tmp0 * (x[index+col]-mean_row);
			    dmean_row -= tmp0;
			}
			dvariance_row *= (-0.5f * powf(variance_row + epsilon, -1.5f));
			dmean_row /= volatility_row;
			for (int col = 0; col < cols; ++col)
			{
			    dmean_row += dvariance_row*(x[index+col] -mean_row) * (-2.0f/cols);
			}
			for (int col = 0; col < cols; ++col)
			{
			    dx[index+col] = (dy[index+col] * gammas[col]) /volatility_row
			                + dvariance_row * (2.0f / cols) * (x[index+col] - mean_row)
			                + dmean_row / cols;
			}
		}
	}


	// for each row of tensor 'x' (of shape (rows, cols), compute the associate mean and variance
	// and stores it in tensor mean and variance (both of shape (rows,1))
	__global__ void Compute_Row_Mean_Variance(int N, int cols, const float* __restrict x, float* __restrict mean, float* __restrict variance, bool unbiasedVariance) 
	{
		int rowId = blockIdx.x * blockDim.x + threadIdx.x;
		if (rowId < N) {
				int startIndex = rowId * cols;
                int endIndex = startIndex + cols - 1;
                double sum = 0.0;
                double sumSquare = 0.0;
                for (int i = startIndex; i <= endIndex; ++i)
                {
					double x_i = x[i];
                    sum += x_i;
                    sumSquare += x_i * x_i;
                }
                float row_mean = (float)(sum / cols);
                mean[rowId] = row_mean;
                int divider = unbiasedVariance ? (cols - 1) : cols;
                float row_variance = (float) abs(sumSquare - cols * row_mean * row_mean) / divider;
                variance[rowId] = row_variance;
		}
	}


	__global__ void ComputeAccuracy(int N, int categoryCount, float *countOk, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted) 
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			if (categoryCount == 1)
			{
				float error = fabsf(yExpectedOneHot[i] - yPredicted[i]);
				countOk[i] = (error < 0.5f) ? 1.0f : 0.0f;
				return;
			}

			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			int maxIndexPredicted = startIndex;
			int maxIndexExpected = startIndex;
			for (int j = startIndex+1; j < endIndexExcluded; ++j)
			{
				if (yPredicted[j] > yPredicted[maxIndexPredicted])
					maxIndexPredicted = j;
				if (yExpectedOneHot[j] > yExpectedOneHot[maxIndexExpected])
					maxIndexExpected = j;
			}
			countOk[i] = (maxIndexPredicted == maxIndexExpected) ? 1.0f : 0.0f;
		}
	}



	__device__  bool IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(const float* __restrict expected, const float* __restrict predicted, int endIndexExcluded, int *pNexIndexToCheck, int subCategoriesCount)
	{
		int subCategoriesFound = 0;
		int predictedSubCategoryId = -1;
		float bestPredictedSubCategoryProba = -1.0f;
		int expectedSubCategoryId = -1;
		float bestExpectedSubCategoryProba = -1.0f;
		bool isAccurate = true;
		bool previousIndexWasProba = false;

		while (subCategoriesFound < subCategoriesCount && (*pNexIndexToCheck < endIndexExcluded))
		{
			float expectedProba = expected[*pNexIndexToCheck];
			float predictedProba = predicted[*pNexIndexToCheck];
			if (fabsf(expectedProba) < 9.5f)
			{
				previousIndexWasProba = true;
				++subCategoriesFound;
				if (expectedProba > bestExpectedSubCategoryProba)
				{
					bestExpectedSubCategoryProba = expectedProba;
					expectedSubCategoryId = subCategoriesFound - 1;
				}
				if (predictedProba > bestPredictedSubCategoryProba)
				{
					bestPredictedSubCategoryProba = predictedProba;
					predictedSubCategoryId = subCategoriesFound - 1;
				}
				*pNexIndexToCheck += 1;
			}
			else
			{
				int count = (int)(fabsf(expectedProba) + 0.5f) / 10;
				if (expectedProba < 0)
				{
					//we need to skip 'count' indexes
					*pNexIndexToCheck += count;
				}
				else
				{
					*pNexIndexToCheck += 1;
					bool subCategoryIsAccurate = IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(expected, predicted, endIndexExcluded, pNexIndexToCheck, count);
					isAccurate = subCategoryIsAccurate && isAccurate;
				}
				if (!previousIndexWasProba)
				{
					++subCategoriesFound;
				}
				previousIndexWasProba = false;
			}
		}
		return (expectedSubCategoryId == predictedSubCategoryId) && isAccurate;
	}

	__device__ inline float IsCountAssociateWithAboveProba(float f) { return f > 5.0f && ((int)(f + 0.1f)) % 10 == 1; }
	__device__ inline float IsProba(float f) { return fabsf(f) < 5.0f; }
	__device__ inline float ExtractCount(float f) { return (int)(fabsf(f) + 0.5f) / 10; }

	__global__ void ComputeSingleAccuracyForCategoricalCrossentropyWithHierarchy(int N, int nbCols, float* countOk, const float* __restrict expected, const float* __restrict predicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int nexIndexToCheck = 0;
			countOk[i] = IsAccuratePredictionForCategoricalCrossentropyWithHierarchy(expected + i * nbCols, predicted + i * nbCols, nbCols, &nexIndexToCheck, 10000000)
				? 1.0f 
				: 0.0f;
		}
	}

	__device__  void SoftmaxWithHierarchy(const float* activationParameter, float* y, int endIndexExcluded, int* pNexIndexToCheck)
	{
		float param = activationParameter[*pNexIndexToCheck];
		y[*pNexIndexToCheck] = param;
		int subCategoriesCount = ExtractCount(param);
		*pNexIndexToCheck += 1;
		
		//we only allocate an array if we have more then '10' elements
		int smallIntArray[10];
		int* indexesProba = (subCategoriesCount > 10) ? (int*)malloc(subCategoriesCount * sizeof(int)) : (&smallIntArray[0]);

		float maxProba = -1e9f;
		bool probaFound = false;

		for (int subCategoriesFound = 0; subCategoriesFound < subCategoriesCount; ++subCategoriesFound)
		{
			float expectedProba = activationParameter[*pNexIndexToCheck];
			if (IsProba(expectedProba))
			{
				maxProba = fmaxf(maxProba, y[*pNexIndexToCheck]);
				indexesProba[subCategoriesFound] = *pNexIndexToCheck;
				probaFound = true;
				*pNexIndexToCheck += 1;
				if (*pNexIndexToCheck < endIndexExcluded && IsCountAssociateWithAboveProba(activationParameter[*pNexIndexToCheck]))
				{
					SoftmaxWithHierarchy(activationParameter, y, endIndexExcluded, pNexIndexToCheck);
				}
			}
			else
			{
				SoftmaxWithHierarchy(activationParameter, y, endIndexExcluded, pNexIndexToCheck);
			}
		}

		if (probaFound)
		{
			float sumExp = 0.0f;
			for (int i = 0; i < subCategoriesCount; ++i)
			{
				int idx = indexesProba[i];
				float tmp = expf(y[idx] - maxProba);
				sumExp += tmp;
				y[idx] = tmp;
			}
			for (int i = 0; i < subCategoriesCount; ++i)
			{
				y[indexesProba[i]] /= sumExp;
			}
		}

		if (subCategoriesCount > 10)
		{
			free(indexesProba);
		}
	}

	__global__ void ComputeSoftmaxWithHierarchy(int N, int nbCols, const float* activationParameter, float* y)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int nexIndexToCheck = 0;
			SoftmaxWithHierarchy(activationParameter, y + i * nbCols, nbCols, &nexIndexToCheck);
		}
	}

	__global__ void ComputeSoftmaxGradientWitHierarchy(int N, int nbCols, const float* activationParameter, const float* y, const float* dy, float* dx)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float expectedProba = activationParameter[i%nbCols];
			if (IsProba(expectedProba))
			{
				float dyi = dy[i];
				float yi = y[i];
				dx[i] = (fabsf(dyi - 1.0f) < 1e-6) ? (yi * (1 - yi)) : (-yi * dyi);
			}
			else
			{
				dx[i] = expectedProba;
			}
		}
	}

	__global__ void SwishGradient(int N, const float* __restrict Y, const float* __restrict dY, const float* __restrict X, float *dX) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < N) {
			float x = X[row];
		    float sigmoid_x = (fabs(x) < 0.0001f) ? 0.5f : Y[row] / x;
            dX[row] = dY[row] * (sigmoid_x + x * sigmoid_x * (1 - sigmoid_x));
		}
	}


	__global__ void ComputeLn(int N, const float* __restrict X, float* Y)
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < N) {
			float x = X[row];
			Y[row] = (x<=0) ? (-100.0f) : logf(x);
		}
	}

	__global__ void LnGradient(int N, const float* __restrict dY, const float* __restrict X, float* dX)
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < N) {
			float x = X[row];
			dX[row] = dY[row] / x;
		}
	}

	__global__ void Set1InMainDiagonal(int nbRows, int nbCols, float *result) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows && row < nbCols) {
			result[row*nbCols+row] = 1.0f;
		}
	}

	__global__ void SetToZeroAllElementsBelowMainDiagonal(int nbRows, int nbCols, float *result) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows) {
			result += row*nbCols;
			int last_i = min(row, nbCols);
			for(int i=0;i<last_i;++i)
			{
				result[i] = 0.0f;
			}
		}
	}

	__global__ void MultiplyEachRowIntoSingleValue(int nbRows, int nbCols, float *result, const float* __restrict a, const float* __restrict b) 
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows) {
			a += row*nbCols;
			b += row*nbCols;
			float sumInRow = 0;
			for(int i=0;i<nbCols;++i)
			{
				sumInRow += (*a)*(*b);
				++a;
				++b;
			}
			result[row] = sumInRow;
		}
	}

	__global__ void Clip(int nbRows, int nbCols, float* result, float lower, float upper)
	{
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		if (row < nbRows) {
			result += row * nbCols;
			for (int i = 0; i < nbCols; ++i)
			{
				result[i] = fminf(fmaxf(result[i], lower), upper);
			}
		}
	}

	// src tensor (unpadded tensor) has shape (n, c, h_src, w_src)
	// dest tensor (padded tensor) has shape (n, c, h_dest, w_dest) with:
    //		h_dest = top_pad + h_src + bottom_pad;
    //      w_dest = left_pad + w_src + right_pad;
	// N = n*c*h_src = number of distinct rows in 'src' tensor
	__global__ void ApplyZeroPaddingForRowId(int N, int h_src, int w_src, int top_pad, int bottom_pad, int left_pad, int right_pad, float* paddedTensor, float* unpaddedTensor, bool isUnpadding) 
	{
		// 'rowId' is the index of the row in 'src' tensor (0 <= rowId < N with N=n*c*h_src)
		int rowId = blockIdx.x * blockDim.x + threadIdx.x;
		if (rowId < N) {
			//we'll copy the row 'rowId' from 'src' tensor (n, c, h_src, w_src) to 'dest' tensor (n, c, h_dest, w_dest)
            int h_dest = top_pad + h_src + bottom_pad;
            int w_dest = left_pad + w_src + right_pad;
            int row_in = (rowId % h_src);
            int destRowIdx = ((rowId / h_src) * h_dest + row_in + top_pad) * w_dest + left_pad;
            int rowIdx = rowId * w_src;
			if (isUnpadding)
				memcpy(unpaddedTensor+rowIdx, paddedTensor+destRowIdx, sizeof(float)*w_src);
			else
				memcpy(paddedTensor+destRowIdx, unpaddedTensor+rowIdx, sizeof(float)*w_src);
		}
	}


	//'x' shape:                (batchSize, timeSteps, inputSize)
	//'y' shape :               (batchSize, timeSteps, outputSize)
	//'wordEmbedding' shape:    (vocabularySize, embeddingDim)
	__global__ void WordEmbeddingForwardPropagation(int N, int inputSize, int outputSize, int xIndexInLastDimensionToUse, int yIndexInLastDimensionToUse, int copyCountBeforeIndex, int copyCountAfterIndex, int embeddingDim, float* x, float* y, float* wordEmbedding)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;	// in [0, batchSize*timeSteps-1]
		if (i >= N) return;

		int xIndex = xIndexInLastDimensionToUse + i * inputSize;
		int yIndex = yIndexInLastDimensionToUse + i * outputSize;
		//we retrieve the wordIndex 
		int wordIndex = (int)(x[xIndex] + 0.1f);	//in [0, vocabularySize-1]
		int indexInWordEmbedding = wordIndex*embeddingDim;

		//for the current timeStep, we copy the elements from 'x' to 'y' before 'indexInLastDimensionToUse'
		if (copyCountBeforeIndex > 0)
		{
			memcpy(y + yIndex-copyCountBeforeIndex, x + xIndex-copyCountBeforeIndex, sizeof(float) * copyCountBeforeIndex);
		}

		if (embeddingDim>0)
		{
			memcpy(y+ yIndex, wordEmbedding+indexInWordEmbedding, sizeof(float) * embeddingDim);
		}

		//for the current timeStep, we copy the elements from 'x' to 'y' after 'indexInLastDimensionToUse'
		if (copyCountAfterIndex > 0)
		{
			memcpy(y + yIndex+ embeddingDim, x + xIndex+1, sizeof(float) * copyCountAfterIndex);
		}
	}

	// N :						batchSize * timeSteps
	// 'x' & 'dx' shape :       (batchSize, timeSteps, inputSize)
	// 'dy' shape :             (batchSize, timeSteps, outputSize)
	//'dw' shape:				(vocabularySize, embeddingDim)
	__global__ void WordEmbeddingBackwardPropagation(int N, int inputSize, int outputSize, int xIndexInLastDimensionToUse, int yIndexInLastDimensionToUse, int copyCountBeforeIndex, int copyCountAfterIndex, int embeddingDim, float* x, float* dx, float* dy, float* dw)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;	// in [0, batchSize*timeSteps-1]
		if (i >= N) return;

		int dxIndex = xIndexInLastDimensionToUse + i * inputSize;

		//we retrieve the wordIndex 
		int wordIndex = (int)(x[dxIndex] + 0.1f);	//in [0, vocabularySize-1]
		int indexInDw = embeddingDim * wordIndex;

		int dyIndex = yIndexInLastDimensionToUse + i * outputSize;
		for (int embeddingId = 0; embeddingId < embeddingDim; ++embeddingId)
		{
			float valueToAdd = dy[dyIndex+embeddingId];
			atomicAdd(dw+ indexInDw, valueToAdd);
			++indexInDw;
		}

		//we initialize 'dx' for the current batchIndex & timeStep
		//for the current timeStep, we copy the elements from 'dy' to 'dx' before 'indexInLastDimensionToUse'
		if (copyCountBeforeIndex > 0)
		{
			memcpy(dx + dxIndex-copyCountBeforeIndex, dy + dyIndex-copyCountBeforeIndex, sizeof(float) * copyCountBeforeIndex);
		}
		dx[dxIndex] = 0.0f;
		//for the current timeStep, we copy the elements from 'dy' to 'dx' after 'indexInLastDimensionToUse'
		if (copyCountAfterIndex > 0)
		{
			memcpy(dx + dxIndex+ 1, dy + dyIndex + embeddingDim, sizeof(float) * copyCountAfterIndex);
		}
	}

	__global__ void YOLOV3Forward(int N, float* y, float* x, int x_c, int x_h, int x_w, int inputImageHeight, int inputImageWidth, int anchor0Width, int anchor0Height, int anchor1Width, int anchor1Height, int anchor2Width, int anchor2Height) 
	{
		int xpredictionIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (xpredictionIndex < N) {

			int nbAnchors = 3;
			int predictionLength = x_c/nbAnchors;
            int categories = predictionLength - 5;
            int rowStride = inputImageHeight / x_h;
            int colStride = inputImageWidth / x_w;
			int xpredictionIndexBackup = xpredictionIndex;
			int elementId = xpredictionIndex / (nbAnchors*x_h*x_w);
			xpredictionIndex = xpredictionIndex %(nbAnchors*x_h*x_w);
			int boxId = xpredictionIndex / (x_h*x_w);
			xpredictionIndex = xpredictionIndex %(x_h*x_w);
			int x_row = xpredictionIndex / (x_w);
			int x_col = xpredictionIndex %(x_w);

			xpredictionIndex= xpredictionIndexBackup;
			int xIndex = elementId*x_c*x_h*x_w + boxId*predictionLength*x_h*x_w + x_row*x_w  + x_col;
			int yIndex = elementId*x_c*x_h*x_w + x_row*x_c*x_w + x_col*x_c + boxId*predictionLength;

            //box center
            y[yIndex++] = (x_col + sigmoidf(x[xIndex])) * colStride;
            xIndex += x_h*x_w;
            y[yIndex++] = (x_row + sigmoidf(x[xIndex])) * rowStride;
            xIndex += x_h*x_w;

            //box size
            int anchorWidth = (boxId == 0) ? anchor0Width : ((boxId == 1) ? anchor1Width : anchor2Width);
            y[yIndex++] = anchorWidth * expf(x[xIndex]);
            xIndex += x_h*x_w;
            int anchorHeight = (boxId == 0) ? anchor0Height : ((boxId == 1) ? anchor1Height : anchor2Height);
            y[yIndex++] = anchorHeight * expf(x[xIndex]);
            xIndex += x_h*x_w;

            //box confidence
            y[yIndex++] = sigmoidf(x[xIndex]);
            xIndex += x_h*x_w;

            //categories
            for (int i = 0; i < categories; ++i)
            {
                y[yIndex++] = sigmoidf(x[xIndex]);
                xIndex += x_h*x_w;
            }
		}
	}


	// src tensor (tensor before up sampling) has shape (n, c, h_src, w_src)
	// dest tensor (tensor after upsampling) has shape (n, c, rowFactor*h_src, colFactor*w_dest)
	// isUpscaling : true if we are up sampling (from 'src' to 'dest') / false if we are down sampling (from 'dest' to 'src')
	__global__ void UpSampling2D(int N, int channels, int h_src, int w_src, int rowFactor, int colFactor, float* src, float* dest, bool isUpscaling) 
	{
		int srcIndex = blockIdx.x * blockDim.x + threadIdx.x;
		if (srcIndex < N) {
			int h_dest = h_src * rowFactor;
			int w_dest = w_src * colFactor;
			float originalElement = src[srcIndex];
			int srcIndexbackup = srcIndex;
	
			int elementId = srcIndex / (channels*h_src*w_src);
			srcIndex = srcIndex %(channels*h_src*w_src);
			int channel = srcIndex / (h_src*w_src);
			srcIndex = srcIndex %(h_src*w_src);
			int row_src = srcIndex / (w_src);
			int col_src = srcIndex %(w_src);
			srcIndex = srcIndexbackup;
			float sum = 0; //only used when down sampling (isUpscaling = false)

			int startOfRow = elementId*(channels*h_dest*w_dest)+channel*(h_dest*w_dest)+ row_src*rowFactor *w_dest + col_src* colFactor;
			for(int rowOffset=0;rowOffset<rowFactor;++rowOffset)
			{
				int idx_dest = startOfRow;
				for(int colOffset=0;colOffset<colFactor;++colOffset)
				{
					if (isUpscaling)
						dest[idx_dest] = originalElement;
					else
						sum += dest[idx_dest];
					++idx_dest;
				}
				startOfRow += w_dest;
			}
			if (!isUpscaling)
				src[srcIndex] = sum;
		}
	}

	__global__ void BinaryCrossentropyLoss(int N, int categoryCount, float* losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				//if ((predicted>0.01)&&(predicted<0.99f))
				if ((predicted > 0.0f) && (predicted < 1.0f))
					loss -= (expected * logf(predicted) + (1.0f - expected) * logf(1.0f - predicted)) / categoryCount;
			}
			losses[i] = loss;
		}
	}

	__global__ void CategoricalCrossentropyLoss(int N, int categoryCount, float *losses, const float* __restrict yExpectedOneHot, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * categoryCount;
			int endIndexExcluded = startIndex + categoryCount;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float predicted = yPredicted[j];
				float expected = yExpectedOneHot[j];
				if (predicted > 0)
					loss -= expected * logf(predicted);
			}
			losses[i] = loss;
		}
	}

	__global__ void CategoricalCrossentropyWithHierarchyLoss(int N, int nbCols, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float loss = 0.0f;
			int startIndex = i * nbCols;
			int endIndexExcluded = startIndex + nbCols;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float expected = yExpected[j];
				if (fabsf(expected) < 9.5f)
				{
					if (expected > 1e-6f)
					{
						//expected contains a proba between 0 and 1
						float predicted = yPredicted[j];
						loss += expected * logf(fmaxf(1e-6f, predicted));
					}
				}
				else
				{
					if (expected < 0) 
					{
						//expected contains a description : there is no associated loss
						int count = (int)(fabsf(expected) + 0.5f) / 10;
						//we need to skip 'count' indexes
						j += count - 1; //-1 because the for(;;) loop will also increment 'j'
					}
				}
			}
			losses[i] = -loss;
		}
	}

	__global__ void CategoricalCrossentropyWithHierarchyGradient(int N, int nbCols, float* loss, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			int startIndex = i * nbCols;
			int endIndexExcluded = startIndex + nbCols;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float expected = yExpected[j];
				if (fabsf(expected) < 9.5f)
				{
					//expected contains a proba between 0 and 1
					loss[j] = yPredicted[j] - expected;
				}
				else
				{
					if (expected < 0)
					{
						//expected contains a number of element to skip: there is no associated loss
						int count = (int)(fabsf(expected) + 0.5f) / 10;
						//we need to skip 'count' indexes
						j += count - 1; //-1 because the for(;;) loop will also increment 'j'
					}
				}
			}
		}
	}

	__global__ void HuberLoss(int batchSize, int lineSize, float huberDelta, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			float loss = 0.0f;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yExpected[j] - yPredicted[j];
				if (fabsf(diff) <= huberDelta)
					loss += 0.5f * diff * diff;
				else
					loss += huberDelta * fabs(diff) - 0.5f * huberDelta * huberDelta;
			}
			losses[i] = loss;
		}
	}

	__global__ void HuberGradient(int batchSize, int lineSize, float huberDelta, float* huberGradient, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yPredicted[j] - yExpected[j];
				huberGradient[j] = fmaxf(fminf(diff, huberDelta), -huberDelta) / lineSize;
			}
		}
	}

	__global__ void MseLoss(int batchSize, int lineSize, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			float loss = 0.0f;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yExpected[j] - yPredicted[j];
				loss += diff * diff;
			}
			losses[i] = loss / lineSize;
		}
	}
	
	__global__ void MseGradient(int batchSize, int lineSize, float* mseGradient, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yPredicted[j] - yExpected[j];
				mseGradient[j] = (2 * diff) / lineSize;
			}
		}
	}

	__global__ void CosineSimilarityLoss(int timeSeriesLength, int yExpectedLength, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int day = blockIdx.x * blockDim.x + threadIdx.x;
		if (day < timeSeriesLength) {
			float top = 0.0f;
            float expectedSquares = 0.0f;
            float predictedSquares = 0.0f;
            for (int t = day; t < yExpectedLength; t+= timeSeriesLength)
            {
                float pred = yPredicted[t];
                float exp = yExpected[t];
                top += pred * exp;
                expectedSquares += exp * exp;
                predictedSquares += pred * pred;
            }
            float l2_norm_expected = sqrtf(expectedSquares);
            float l2_norm_predicted = sqrtf(predictedSquares);
            losses[day] = top / (l2_norm_expected * l2_norm_predicted);
		}
	}

	__global__ void CosineSimilarityGradient(int timeSeriesLength, int yExpectedLength, float* cosineSimilarityGradient, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int day = blockIdx.x * blockDim.x + threadIdx.x;
		if (day < timeSeriesLength) {
			double top = 0.0;
            double expectedSquares = 0.0;
            double predictedSquares = 0.0;
            for (int t = day; t < yExpectedLength; t+= timeSeriesLength)
            {
                double pred = yPredicted[t];
                double exp = yExpected[t];
                top += pred * exp;
                expectedSquares += exp * exp;
                predictedSquares += pred * pred;
            }
            double l2_norm_expected = sqrt(expectedSquares);
            double l2_norm_predicted = sqrt(predictedSquares);
            double multiplier1 = 1.0/(l2_norm_expected * l2_norm_predicted);
            double mutliplier2 = (-top)/(l2_norm_predicted* l2_norm_predicted * l2_norm_predicted * l2_norm_expected);
            for (int t = day; t < yExpectedLength; t += timeSeriesLength)
            {
                cosineSimilarityGradient[t] = - (float)(multiplier1*yExpected[t] + mutliplier2*yPredicted[t]);
            }
		}
	}

	__global__ void MeanSquaredLogErrorLoss(int batchSize, int lineSize, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			float loss = 0.0f;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = logf(1+yPredicted[j]) - logf(1+yExpected[j]);
				loss += diff * diff;
			}
			losses[i] = loss / lineSize;
		}
	}

	__global__ void MseOfLogLoss(int batchSize, int lineSize, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted, float epsilon)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			float loss = 0.0f;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float adjustedPredicted = fmaxf(epsilon, yPredicted[j]);
				float diff = logf(adjustedPredicted) - logf(yExpected[j]);
				loss += diff * diff;
			}
			losses[i] = loss / lineSize;
		}
	}


	__global__ void MseOfLogGradient(int batchSize, int lineSize, float* mseGradient, const float* __restrict yExpected, const float* __restrict yPredicted, float epsilon)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float adjustedPredicted = fmaxf(epsilon, yPredicted[j]);
				float diff = logf(adjustedPredicted) - logf(yExpected[j]);
				mseGradient[j] = (2 * diff) / (adjustedPredicted*lineSize);
			}
		}
	}


	__global__ void MaeLoss(int batchSize, int lineSize, float* losses, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			float loss = 0.0f;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yExpected[j] - yPredicted[j];
				loss += fabsf(diff);
			}
			losses[i] = loss / lineSize;
		}
	}

	__global__ void MaeGradient(int batchSize, int lineSize, float* mseGradient, const float* __restrict yExpected, const float* __restrict yPredicted)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < batchSize) {
			int startIndex = i * lineSize;
			int endIndexExcluded = startIndex + lineSize;
			for (int j = startIndex; j < endIndexExcluded; ++j)
			{
				float diff = yPredicted[j] - yExpected[j];
				mseGradient[j] = (diff>=0?1.0f:-1.f) / lineSize;
			}
		}
	}


	// Compute:  y = slope * x + intercept
	__global__ void LinearFunction(int N, float* y, float slope, const float* x, float intercept)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N)
		{
			y[i] = slope * x[i] + intercept;
		}
	}

	__global__ void Concatenate(int N, int m, float* __restrict concat, int concatMultDim0, const float* __restrict a, int aMultDim0, const float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			concat[i] = a[row*aMultDim0+colInConcat];
		else
			concat[i] = b[row*bMultDim0+colInConcat-aMultDim0];

	}

	__global__ void Concatenate3(int N, int m, float* __restrict concat, int concatMultDim0, const float* __restrict a, int aMultDim0, const float* __restrict b, int bMultDim0, const float* __restrict c, int cMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			concat[i] = a[row*aMultDim0+colInConcat];
		else
			concat[i] = (colInConcat<(aMultDim0+bMultDim0))?b[row*bMultDim0+colInConcat-aMultDim0]:c[row*cMultDim0+colInConcat-aMultDim0-bMultDim0];
	}


	__global__ void Split(int N, int m, const float* __restrict concat, int concatMultDim0, float* __restrict a, int aMultDim0, float* __restrict b, int bMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
	}

	__global__ void Split3(int N, int m, const float* __restrict concat, int concatMultDim0, float* __restrict a, int aMultDim0, float* __restrict b, int bMultDim0, float* __restrict c, int cMultDim0)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)  return;
		int row = i/concatMultDim0;
		int colInConcat = i%concatMultDim0;
		if (colInConcat<aMultDim0)
			a[row*aMultDim0+colInConcat] = concat[i];
		else if (colInConcat<(aMultDim0+bMultDim0))
			b[row*bMultDim0+colInConcat-aMultDim0] = concat[i];
		else
			c[row*cMultDim0+colInConcat-aMultDim0-bMultDim0] = concat[i];
	}

	// transform a 'src' tensor of shape [a,b,c] to a 'target' tensor of shape [b,a,c] 
	__global__ void Switch_First_2_axis(int N, int aLength, int bLength, int cLength, const float* __restrict src, float *target)
	{
		int idx_src = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx_src < N) {
			int multDim0 = bLength * cLength;
			int a_src = idx_src / multDim0;
			int tmp = idx_src % multDim0;
			int b_src = tmp / cLength;
			int c_src = tmp % cLength;
			int idx_target = b_src * aLength * cLength + a_src * cLength + c_src;
			target[idx_target] = src[idx_src];
		}
	}

	// transform a 'src' tensor of shape [n,c,h] to a 'target' tensor of shape [n,h,c] 
	__global__ void SwitchSecondAndThirdDimension(int N, int nLength, int cLength, int hLength, const float* __restrict src, float* target)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N)
			return;
		int n = i / cLength;
		int c = i % cLength;
		for(int h=0;h<hLength;++h)
		{
			int src_index = h + c * hLength + n * cLength * hLength;
			int target_index = c + h * cLength + n * cLength * hLength;
			target[target_index] = src[src_index];
		}
	}
}

